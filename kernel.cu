#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <vector>
#include <iterator>
#include <fstream>
#include <algorithm>
#include "gputimer.h"

#define NUMX 256
#define NUMY 256
#define STRAIN 0.00
#define POISSON 0.165
#define A0 1.42
#define NUM_THREADS 262144
#define BLOCK_WIDTH 1024
#define RESOLUTION 100
#define ZRES 100
#define SMALL_Z 0.00001
#define Z_MAX 20.0



struct atom_t {
	float atompos, vLJ;

	friend std::istream& operator >> (std::istream& ins, atom_t& r);
};

std::istream& operator >> (std::istream& ins, atom_t& r) {
	ins >> r.atompos >> r.vLJ;
	return ins;
};

template<class T>
void print_array(T *array, int size)
{
	printf("{ ");
	for (int i = 0; i < size; i++)  { printf("%e ", array[i]); }
	printf("}\n");
}

__global__ void potentialLJ(
	float epsilon,
	float sigma,
	int x,
	int y,
	int z,
	float *pos_carbonx,
	float *pos_carbony,
	float *pos_atomx,
	float *pos_atomy,
	float *pos_atomz,
	float *potLJ,
	int numCarbons) 
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCarbons + thread_2D_pos.x;
	float cx = pos_carbonx[thread_1D_pos];
	float cy = pos_carbony[thread_1D_pos];

	float rx = pos_atomx[x] - cx;
	float ry = pos_atomy[y] - cy;
	float rz = pos_atomz[z];
	float invr = rsqrtf(rx * rx + ry * ry + rz * rz);
	float sor6 = sigma * sigma * sigma * sigma * sigma * sigma * invr * invr * invr * invr * invr * invr;
	float vLJ = 4.0 * epsilon * sor6 * (sor6 - 1.0);
	// accumulate effect of all particles
	atomicAdd(potLJ, vLJ);
	
}

int main(int argc, char **argv)
{
	GpuTimer timer;
	// declare variables
	int numCarbons = NUM_THREADS;
	int numx = NUMX;
	int numy = NUMY;
	
	float epsilon = 16.2463;
	float sigma = 2.74;
	float smallz = SMALL_Z;
	float zmax = Z_MAX;
	//////////////////////////////////////////////////////////////////////////////	
	// Create graphene lattice
	//////////////////////////////////////////////////////////////////////////////
	float ax = A0 * (1.00 + (STRAIN * (1.0 - (3.0 * POISSON)) / 4.0)); // New Way
	float ay = A0 * (1.00 + STRAIN);
	float ay2 = (3.0/8.0)*A0*(4.0+(3.0*STRAIN)-(STRAIN*POISSON));
	float d0 = sqrt((float)3.0) * ax;
	// Create a vector object that contains numx elements.
	std::vector<float> transx;
	for (int i = 0; i < numx; ++i) {
		transx.push_back((i - numx / 2)*d0);
	}
	std::vector<float> transx2;
	for (int i = 0; i < numx; ++i) {
		transx2.push_back((d0 / 2) + (i - numx / 2)*d0);
	}
	std::vector<float> x(((transx.size() + transx2.size()) * 2 * numy));
	for (int k = 0; k < numy; ++k) {
		for (int j = 0; j < numy; ++j) {
			for (int i = k * 4 * numy; i < x.size(); i += 4) {
				x[i] = transx[k];
				x[i + 1] = transx[k];
				x[i + 2] = transx2[k];
				x[i + 3] = transx2[k];
			}
		}
	}
	// Create a vector object that contains numy elements.
	std::vector<float> transy;
	for (int i = 0; i < numy; ++i) {
		transy.push_back((((ay + ax) / 2) + (ax + ay + ay)*(i - numy / 2)));
	}
	std::vector<float> transy2;
	for (int i = 0; i < numy; ++i) {
		transy2.push_back(((-(ay + ax) / 2) + (ax + ay + ay)*(i - numy / 2)));
	}
	std::vector<float> transy3;
	for (int i = 0; i < numy; ++i) {
		transy3.push_back(((ay / 2) + (ax + ay + ay)*(i - numy / 2)));
	}
	std::vector<float> transy4;
	for (int i = 0; i < numy; ++i) {
		transy4.push_back(((-ay / 2) + (ax + ay + ay)*(i - numy / 2)));
	}
	std::vector<float> tempy(4 * numx);
	for (int i = 0, k = 0; k < numy; i += 4, ++k) {
		tempy[i] = transy[k];
		tempy[i + 1] = transy2[k];
		tempy[i + 2] = transy3[k];
		tempy[i + 3] = transy4[k];
	}
	std::vector<float> y;
	for (int i = 0; i < numx; ++i) {
		copy(tempy.begin(), tempy.end(), back_inserter(y));
	}




	//////////////////////////////////////////////////////////////////////////////	
	// Test positions
	//////////////////////////////////////////////////////////////////////////////

	std::vector<float> pos_atomx;
	for (int i = 0; i < RESOLUTION + 1; ++i) {
		pos_atomx.push_back(i*d0/(2*(RESOLUTION)));
	}

	std::vector<float> pos_atomy;
	for (int i = 0; i < RESOLUTION + 1; ++i) {
		pos_atomy.push_back(i*ay2/(RESOLUTION));
	}

	std::vector<float> pos_atomz;
	for (int i = 0; i < ZRES + 1; ++i) {
		pos_atomz.push_back(smallz + i*zmax/(RESOLUTION));
	}


	//////////////////////////////////////////////////////////////////////////////
	// declare and allocate host memory
	//////////////////////////////////////////////////////////////////////////////
	float* h_pos_carbonx = &x[0];
	float* h_pos_carbony = &y[0];
	float* h_pos_atomx = &pos_atomx[0];
	float* h_pos_atomy = &pos_atomy[0];
	float* h_pos_atomz = &pos_atomz[0];
	float h_potLJ[RESOLUTION*RESOLUTION*ZRES];
	float h_potLJtemp[0];
	const int CARBON_BYTES = NUM_THREADS * sizeof(float);
	const int RES_BYTES = RESOLUTION * sizeof(float);
	const int ZRES_BYTES = ZRES * sizeof(float);
	const int POTLJ_BYTES = RESOLUTION * RESOLUTION * ZRES * sizeof(float);
	

	memset(h_potLJ, 0.0, POTLJ_BYTES);
	memset(h_potLJtemp, 0.0, sizeof(float));


	// declare, allocate, and zero out GPU memory
	float *d_pos_carbonx;
	float *d_pos_carbony;
	float *d_pos_atomx;
	float *d_pos_atomy;
	float *d_pos_atomz;
	float *d_potLJ;
	hipMalloc((void **)&d_pos_carbonx, CARBON_BYTES);
	hipMalloc((void **)&d_pos_carbony, CARBON_BYTES);
	hipMalloc((void **)&d_pos_atomx, RES_BYTES);
	hipMalloc((void **)&d_pos_atomy, RES_BYTES);
	hipMalloc((void **)&d_pos_atomz, ZRES_BYTES);
	hipMalloc((void **)&d_potLJ, sizeof(float));

	// now copy data from host memory to device memory
	hipMemcpy((void *)d_pos_carbonx, (void *)h_pos_carbonx, CARBON_BYTES, hipMemcpyHostToDevice);
	hipMemcpy((void *)d_pos_carbony, (void *)h_pos_carbony, CARBON_BYTES, hipMemcpyHostToDevice);
	hipMemcpy((void *)d_pos_atomx, (void *)h_pos_atomx, RES_BYTES, hipMemcpyHostToDevice);
	hipMemcpy((void *)d_pos_atomy, (void *)h_pos_atomy, RES_BYTES, hipMemcpyHostToDevice);
	hipMemcpy((void *)d_pos_atomz, (void *)h_pos_atomz, ZRES_BYTES, hipMemcpyHostToDevice);
	hipMemcpy((void *)d_potLJ, (void *)h_potLJtemp, sizeof(float), hipMemcpyHostToDevice);


	//////////////////////////////////////////////////////////////////////////////
	// launch the kernel
	//////////////////////////////////////////////////////////////////////////////
	printf("%d total threads in %d blocks writing into %d array elements\n",
		NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, NUM_THREADS);	
	timer.Start();
	
	for (int i = 0; i < RESOLUTION; ++i) {
		for (int j = 0; j < RESOLUTION; ++j){
			for (int k = 0; k < ZRES; ++k){

	potentialLJ << <NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >> >(
		epsilon,
		sigma,
		i,
		j,
		k,
		d_pos_carbonx,
		d_pos_carbony,
		d_pos_atomx,
		d_pos_atomy,
		d_pos_atomz,
		d_potLJ,
		numCarbons);
	

	// copy back the array of sums from GPU and print
	hipMemcpy(h_potLJtemp, d_potLJ, sizeof(float), hipMemcpyDeviceToHost);
	h_potLJ[i * RESOLUTION * ZRES + j * ZRES + k] = h_potLJtemp[0];

	memset(h_potLJtemp, 0.0, sizeof(float));
	hipMemcpy((void *)d_potLJ, (void *)h_potLJtemp, sizeof(float), hipMemcpyHostToDevice);
			}
		}
	}


	// End LOOP

	
	//std::ofstream xout("carbonx.txt");
	//for (int i = 0; i < numCarbons; i++)
	//{
	//	xout << h_pos_carbonx[i] << std::endl; //writing ith character of array in the file
	//}
	//std::ofstream yout("carbony.txt");
	//for (int i = 0; i < numCarbons; i++)
	//{
	//	yout << h_pos_carbony[i] << std::endl; //writing ith character of array in the file
	//}

	std::ofstream fout("out.txt");
	for (int i = 0; i < RESOLUTION*RESOLUTION*ZRES; i++)
	{
		fout << h_potLJ[i] << std::endl; //writing ith character of array in the file
	}
	timer.Stop();
	

	printf("Time elapsed = %g ms\n", timer.Elapsed());
	//print_array(&pos_atomx[0], RESOLUTION+1);
	//print_array(&pos_atomy[0], RESOLUTION+1);
	//print_array(&pos_atomz[0], ZRES+1);

	// free GPU memory allocation and exit
	hipFree(d_pos_carbonx);
	hipFree(d_pos_carbony);
	hipFree(d_pos_atomx);
	hipFree(d_pos_atomy);
	hipFree(d_pos_atomz);
	hipFree(d_potLJ);
	return 0;
}


